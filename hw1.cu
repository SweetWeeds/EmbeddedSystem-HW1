#include "user.h"
#include "user_host.h"
#include "user_device.h"

using namespace std;

int main(int argc, char *argv[]) {
    random_device rd;
    mt19937 gen(rd());
    uniform_int_distribution<int> dis(1, 10);
    hipEvent_t cuda_start, cuda_end;
    float host_exec_time, device_exec_time;
    
    int mat1_row = MAT1_ROW;
    int mat1_col = MAT1_COL;
    int mat2_row = MAT2_ROW;
    int mat2_col = MAT2_COL;
    int matr_row = MATR_ROW;
    int matr_col = MATR_COL;

    printf("mat1[%d][%d]\n", mat1_row, mat1_col);
    printf("mat2[%d][%d]\n", mat2_row, mat2_col);
    printf("matr[%d][%d]\n", matr_row, matr_col);

    int mat1_size = mat1_row * mat1_col * sizeof(int);
    int mat2_size = mat2_row * mat2_col * sizeof(int);
    int matr_size = matr_row * matr_col * sizeof(int);
    
    int *host_mat1 = NULL;
    int *host_mat2 = NULL;
    int *host_matr = NULL;
    int *device_mat1 = NULL;
    int *device_mat2 = NULL;
    int *device_matr = NULL;
    int *host_device_matr = NULL;
    
    // Start of Memory Allocation //
    host_mat1 = (int *)malloc(mat1_size);
    host_mat2 = (int *)malloc(mat2_size);
    host_matr = (int *)malloc(matr_size);
    host_device_matr = (int *)malloc(matr_size);
    hipMalloc((void **)&device_mat1, mat1_size);
    hipMalloc((void **)&device_mat2, mat2_size);
    hipMalloc((void **)&device_matr, matr_size);
    // End of Memory Allocation //

    // Start of Array Initalization //
    // Matrix 1
    for (int i = 0; i < mat1_row; i++) {
        for (int j = 0; j < mat1_col; j++) {
            host_mat1[i * mat1_col + j] = dis(gen);
        }
    }

    // Matrix 2
    for (int i = 0; i < mat2_row; i++) {
        for (int j = 0; j < mat2_col; j++) {
            host_mat2[i * mat2_col + j] = dis(gen);
        }
    }
    // End of Array Initalization //

    // Memory Copy
    hipMemcpy(device_mat1, host_mat1, mat1_size, hipMemcpyHostToDevice);
    hipMemcpy(device_mat2, host_mat2, mat2_size, hipMemcpyHostToDevice);

    // Prepare File Stream
    ofstream resultFile;
    resultFile.open("./result.csv", ios::out);
    resultFile << "Target,#Threads,#ThreadBlks,ExecTime\n";

    // Start of Concatenation (host) //
    struct timeval startTime, endTime;
    gettimeofday(&startTime, NULL);
    host_Concatenate(host_mat1, host_mat2, host_matr, &host_exec_time);
    gettimeofday(&endTime, NULL);
    host_exec_time = (endTime.tv_sec - startTime.tv_sec) * 1000. + (endTime.tv_usec - startTime.tv_usec) / 1000.;
    resultFile << "host,1,1," << host_exec_time << "\n";
    printf("[INFO] Host Execution time:%lf\n", host_exec_time);
    // End of Concatenation (host) //

    int numElements = matr_col * matr_row;
    int numBlocks, numThreadsperBlock;
    for (numThreadsperBlock = NUM_THREADS_BASE; numThreadsperBlock <= NUM_THREADSA_MAX; numThreadsperBlock *= 2) {
        for (numBlocks = NUM_THREAD_BLKS_FROM; numBlocks <= NUM_THREAD_BLKS_TO; numBlocks *= 2) {
            int numOps = numElements > (numBlocks * numThreadsperBlock) ? numElements / (numBlocks * numThreadsperBlock) + (numElements % (numBlocks * numThreadsperBlock) ? 1 : 0) : 1;
            dim3 gridSize(numBlocks);
            dim3 blockSize(numThreadsperBlock);

            hipMemset(device_matr, 0, numElements);
            hipEventCreate(&cuda_start);
            hipEventCreate(&cuda_end);
            hipEventRecord(cuda_start, 0);
            device_Concatenate<<<gridSize, blockSize>>>(device_mat1, device_mat2, device_matr, numOps, numElements, mat1_col, mat2_col, matr_col);
            hipEventRecord(cuda_end, 0);
            hipEventSynchronize(cuda_end);
            hipEventElapsedTime(&device_exec_time, cuda_start, cuda_end);
            hipEventDestroy(cuda_start);
            hipEventDestroy(cuda_end);

            resultFile << "device," << numThreadsperBlock << "," << numBlocks << "," << device_exec_time << "\n";
            hipMemcpy(host_device_matr, device_matr, matr_size, hipMemcpyDeviceToHost);
            int diff = compareArray(host_matr, host_device_matr, numElements);
            if (diff) {
                printf("[WARNING] ");
            } else {
                printf("[INFO] ");
            }
            printf("numOps: %d, numBlocks: %d, numThreadsperBlock: %d, diff: %d, exec_time: %.4lf\n", numOps, numBlocks, numThreadsperBlock, diff, device_exec_time);
        }
    }

    // Close File Stream
    resultFile.close();

    // End of Concatenation (device) //

    return 0;
}
